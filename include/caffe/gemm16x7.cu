#include "hip/hip_runtime.h"
#ifndef _GEMM16x7_CU_
#define _GEMM16x7_CU_

namespace gemm16x7 {

const int TW_ROW = 16, TW_COL = 7, TW_DEPTH = 16;


#define tile_a_x_tile_b (tile_a[ty][0]*tile_b[0][tx]+tile_a[ty][1]*tile_b[1][tx]+tile_a[ty][2]*tile_b[2][tx]+tile_a[ty][3]*tile_b[3][tx]+ \
  tile_a[ty][4]*tile_b[4][tx]+tile_a[ty][5]*tile_b[5][tx]+tile_a[ty][6]*tile_b[6][tx]+tile_a[ty][7]*tile_b[7][tx]+ \
  tile_a[ty][8]*tile_b[8][tx]+tile_a[ty][9]*tile_b[9][tx]+tile_a[ty][10]*tile_b[10][tx]+tile_a[ty][11]*tile_b[11][tx]+ \
  tile_a[ty][12]*tile_b[12][tx]+tile_a[ty][13]*tile_b[13][tx]+tile_a[ty][14]*tile_b[14][tx]+tile_a[ty][15]*tile_b[15][tx])
 


template <typename Dtype>
__global__ void kernel_gemm_nn(const int M, const int N, const int K,
    const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
    Dtype *c) {
    __shared__ Dtype tile_a[TW_ROW][TW_DEPTH];
    __shared__ Dtype tile_b[TW_DEPTH][TW_COL];

    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int row = blockIdx.y * blockDim.y + ty;
    int col = blockIdx.x * blockDim.x + tx;
    Dtype ans = 0;

    int i;
    for (i = 0; i < K - TW_DEPTH; i += TW_DEPTH)
    {
        // for (int j = 0; j < TW_DEPTH; j += TW_COL) {
        //     if (tx + j < TW_DEPTH) {
                tile_a[ty][tx] = a[row * K + (i + tx)];
                tile_a[ty][tx + TW_COL] = a[row * K + (i + tx + TW_COL)];
                if (tx < 2)
                  tile_a[ty][tx + TW_COL * 2] = a[row * K + (i + tx + TW_COL * 2)];
            // }
        // }
        // for (int j = 0; j < TW_DEPTH; j += TW_ROW) {
        //     if (ty + j < TW_DEPTH) {
                tile_b[ty][tx] = b[(i + ty) * N + col];
            // }
        // }
        __syncthreads();
        ans += tile_a_x_tile_b;
        __syncthreads();
    }

    // for (int j = 0; j < TW_DEPTH; j += TW_COL) {
    //     if (tx + j < TW_DEPTH) {
            tile_a[ty][tx] = (i + tx < K) ? a[row * K + i + tx] : 0;
            tile_a[ty][tx + TW_COL] = (i + tx + TW_COL < K) ? a[row * K + i + tx + TW_COL] : 0;
            if (tx < 2)
            tile_a[ty][tx + TW_COL * 2] = (i + tx + TW_COL * 2 < K) ? a[row * K + i + tx + TW_COL * 2] : 0;
    //     }
    // }
    // for (int j = 0; j < TW_DEPTH; j += TW_ROW) {
    //     if (ty + j < TW_DEPTH) {
            tile_b[ty][tx] = (i + ty < K) ? b[(i + ty) * N + col] : 0;
    //     }
    // }
    __syncthreads();
    ans += tile_a_x_tile_b;
    __syncthreads();  

    ans *= alpha;
    
    if (beta != 0)
      c[row * N + col] = c[row * N + col] * beta + ans;
    else
      c[row * N + col] = ans;
}

template <typename Dtype>
__global__ void kernel_gemm_tn(const int M, const int N, const int K,
  const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
  Dtype *c) {
  __shared__ Dtype tile_a[TW_ROW][TW_DEPTH];
  __shared__ Dtype tile_b[TW_DEPTH][TW_COL];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * blockDim.y + ty;
  int col = blockIdx.x * blockDim.x + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - TW_DEPTH; i += TW_DEPTH)
  {

    // for (int j = 0; j < TW_DEPTH; j += TW_COL)
    //   if (tx + j < TW_DEPTH)
        tile_a[ty][tx] = a[(i + tx) * M + row];
        tile_a[ty][tx + TW_COL] = a[(i + tx + TW_COL) * M + row];
        if (tx < 2)
        tile_a[ty][tx + TW_COL * 2] = a[(i + tx + TW_COL * 2) * M + row];
    // for (int j = 0; j < TW_DEPTH; j += TW_ROW)
    //   if (ty + j < TW_DEPTH)
        tile_b[ty][tx] = b[(i + ty) * N + col];
    __syncthreads();
    ans += tile_a_x_tile_b;
    __syncthreads();
  }

  // for (int j = 0; j < TW_DEPTH; j += TW_COL)
  //     if (tx + j < TW_DEPTH)
          tile_a[ty][tx] = (i + tx < K) ? a[(i + tx) * M + row] : 0;
          tile_a[ty][tx + TW_COL] = (i + tx + TW_COL < K) ? a[(i + tx + TW_COL) * M + row] : 0;
          if (tx < 2)
          tile_a[ty][tx + TW_COL * 2] = (i + tx + TW_COL * 2 < K) ? a[(i + tx + TW_COL * 2) * M + row] : 0;
  // for (int j = 0; j < TW_DEPTH; j += TW_ROW)
  //     if (ty + j < TW_DEPTH)
          tile_b[ty][tx] = (i + ty < K) ? b[(i + ty) * N + col] : 0;
  __syncthreads();
  ans += tile_a_x_tile_b;
  __syncthreads();

  ans *= alpha;
  
  if (beta != 0)
    c[row * N + col] = c[row * N + col] * beta + ans;
  else
    c[row * N + col] = ans;
}

template <typename Dtype>
__global__ void kernel_gemm_nt(const int M, const int N, const int K,
  const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
  Dtype *c) {
  __shared__ Dtype tile_a[TW_ROW][TW_DEPTH];
  __shared__ Dtype tile_b[TW_DEPTH][TW_COL];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * blockDim.y + ty;
  int col = blockIdx.x * blockDim.x + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - TW_DEPTH; i += TW_DEPTH)
  {
    // for (int j = 0; j < TW_DEPTH; j += TW_COL)
    //   if (tx + j < TW_DEPTH)
        tile_a[ty][tx] = a[row * K + i + tx];
        tile_a[ty][tx + TW_COL] = a[row * K + i + tx + TW_COL];
        if (tx < 2)
        tile_a[ty][tx + TW_COL * 2] = a[row * K + i + tx + TW_COL * 2];
    // for (int j = 0; j < TW_DEPTH; j += TW_ROW)
    //   if (ty + j < TW_DEPTH)
        tile_b[ty][tx] = b[col * K + i + ty];
    __syncthreads();
    ans += tile_a_x_tile_b;
    __syncthreads();
  }

  // for (int j = 0; j < TW_DEPTH; j += TW_COL)
  //   if (tx + j < TW_DEPTH)
      tile_a[ty][tx] = (i + tx < K) ? a[row * K + i + tx] : 0;
      tile_a[ty][tx + TW_COL] = (i + tx + TW_COL < K) ? a[row * K + i + tx + TW_COL] : 0;
      if (tx < 2)
      tile_a[ty][tx + TW_COL * 2] = (i + tx + TW_COL * 2 < K) ? a[row * K + i + tx + TW_COL * 2] : 0;
  // for (int j = 0; j < TW_DEPTH; j += TW_ROW)
  //   if (ty + j < TW_DEPTH)
      tile_b[ty][tx] = (i + ty < K) ? b[col * K + i + ty] : 0;
  __syncthreads();
  ans += tile_a_x_tile_b;
  __syncthreads();

  ans *= alpha;
  
  if (beta != 0)
    c[row * N + col] = c[row * N + col] * beta + ans;
  else
    c[row * N + col] = ans;
}

template <typename Dtype>
__global__ void kernel_gemm_tt(const int M, const int N, const int K,
  const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
  Dtype *c) {
  __shared__ Dtype tile_a[TW_ROW][TW_DEPTH];
  __shared__ Dtype tile_b[TW_DEPTH][TW_COL];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * blockDim.y + ty;
  int col = blockIdx.x * blockDim.x + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - TW_DEPTH; i += TW_DEPTH)
  {
    // for (int j = 0; j < TW_DEPTH; j += TW_COL)
    //   if (tx + j < TW_DEPTH)
        tile_a[ty][tx] = a[(i + tx) * M + row];
        tile_a[ty][tx + TW_COL] = a[(i + tx + TW_COL) * M + row];
        if (tx < 2)
        tile_a[ty][tx + TW_COL * 2] = a[(i + tx + TW_COL * 2) * M + row];
    // for (int j = 0; j < TW_DEPTH; j += TW_ROW)
    //   if (ty + j < TW_DEPTH)
        tile_b[ty][tx] = b[col * K + i + ty];
    __syncthreads(); 
    ans += tile_a_x_tile_b;
    __syncthreads();
  }
  // for (int j = 0; j < TW_DEPTH; j += TW_COL)
  //   if (tx + j < TW_DEPTH)
      tile_a[ty][tx] = (i + tx < K) ? a[(i + tx) * M + row] : 0;
      tile_a[ty][tx + TW_COL] = (i + tx + TW_COL < K) ? a[(i + tx + TW_COL) * M + row] : 0;
      if (tx < 2)
      tile_a[ty][tx + TW_COL * 2] = (i + tx + TW_COL * 2 < K) ? a[(i + tx + TW_COL * 2) * M + row] : 0;
  // for (int j = 0; j < TW_DEPTH; j += TW_ROW)
  //   if (ty + j < TW_DEPTH)
      tile_b[ty][tx] = (i + ty < K) ? b[col * K + i + ty] : 0;
  __syncthreads();
  ans += tile_a_x_tile_b;
  __syncthreads();

  ans *= alpha;
  
  if (beta != 0)
    c[row * N + col] = c[row * N + col] * beta + ans;
  else
    c[row * N + col] = ans;
}




// gemm kernel
// 4: 1.3
// 7: 2.7
// 8: 4
// 14: 4.15
// 16: 5.08, pragma unroll(4): 5.18, manually unroll: 5.25
// 18: 4.5
// 20: 4.5
// 32: 4.85
// cublas: 10.5
// gemm interface
void caffe_gpu_gemm(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  int grid_rows = M / TW_ROW;
  int grid_cols = N / TW_COL;
  dim3 gridSize(grid_cols, grid_rows);
  dim3 blockSize(TW_COL, TW_ROW);
  
  if (TransA == CblasNoTrans && TransB == CblasNoTrans)
    kernel_gemm_nn<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);
  else if (TransA != CblasNoTrans && TransB == CblasNoTrans)
    kernel_gemm_tn<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);
  else if (TransA == CblasNoTrans && TransB != CblasNoTrans)
    kernel_gemm_nt<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);
  else
    kernel_gemm_tt<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);

  // hipError_t er1 = hipPeekAtLastError();
  // CUDA_CHECK(er1);
}

}
#endif // _GEMM16x7_CU_