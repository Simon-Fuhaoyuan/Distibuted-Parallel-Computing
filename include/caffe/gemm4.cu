#include "hip/hip_runtime.h"
#ifndef _GEMM4_CU_
#define _GEMM4_CU_

namespace gemm4 {

#define BLOCK_SIZE_4 4
#define taxtb_4 (tile_a[ty][0]*tile_b[0][tx]+tile_a[ty][1]*tile_b[1][tx]+tile_a[ty][2]*tile_b[2][tx]+tile_a[ty][3]*tile_b[3][tx])

template <typename Dtype>
__global__ void kernel_gemm_nn(const int M, const int N, const int K,
    const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
    Dtype *c) {
  __shared__ Dtype tile_a[BLOCK_SIZE_4][BLOCK_SIZE_4];
  __shared__ Dtype tile_b[BLOCK_SIZE_4][BLOCK_SIZE_4];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE_4 + ty;
  int col = blockIdx.x * BLOCK_SIZE_4 + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - BLOCK_SIZE_4; i += BLOCK_SIZE_4)
  {
    tile_a[ty][tx] = (row < M) ? a[row * K + i + tx] : 0;
    tile_b[ty][tx] = (col < N) ? b[(i + ty) * N + col] : 0;
    __syncthreads();
    ans += taxtb_4;
    __syncthreads();
  }

  tile_a[ty][tx] = (row < M && i + tx < K) ? a[row * K + i + tx] : 0;
  tile_b[ty][tx] = (col < N && i + ty < K) ? b[(i + ty) * N + col] : 0;
  __syncthreads();
  ans += taxtb_4;
  __syncthreads();  

  ans *= alpha;
  if (row < M && col < N)
  {
    if (beta != 0)
      c[row * N + col] = c[row * N + col] * beta + ans;
    else
      c[row * N + col] = ans;
  }
}

template <typename Dtype>
__global__ void kernel_gemm_tn(const int M, const int N, const int K,
  const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
  Dtype *c) {
  __shared__ Dtype tile_a[BLOCK_SIZE_4][BLOCK_SIZE_4];
  __shared__ Dtype tile_b[BLOCK_SIZE_4][BLOCK_SIZE_4];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE_4 + ty;
  int col = blockIdx.x * BLOCK_SIZE_4 + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - BLOCK_SIZE_4; i += BLOCK_SIZE_4)
  {
    tile_a[ty][tx] = (row < M) ? a[(i + tx) * M + row] : 0;
    tile_b[ty][tx] = (col < N) ? b[(i + ty) * N + col] : 0;
    __syncthreads();
    ans += taxtb_4;
    __syncthreads();
  }

  tile_a[ty][tx] = (row < M && i + tx < K) ? a[(i + tx) * M + row] : 0;
  tile_b[ty][tx] = (col < N && i + ty < K) ? b[(i + ty) * N + col] : 0;
  __syncthreads();
  ans += taxtb_4;
  __syncthreads();

  ans *= alpha;
  if (row < M && col < N)
  {
    if (beta != 0)
      c[row * N + col] = c[row * N + col] * beta + ans;
    else
      c[row * N + col] = ans;
  }
}

template <typename Dtype>
__global__ void kernel_gemm_nt(const int M, const int N, const int K,
  const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
  Dtype *c) {
  __shared__ Dtype tile_a[BLOCK_SIZE_4][BLOCK_SIZE_4];
  __shared__ Dtype tile_b[BLOCK_SIZE_4][BLOCK_SIZE_4];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE_4 + ty;
  int col = blockIdx.x * BLOCK_SIZE_4 + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - BLOCK_SIZE_4; i += BLOCK_SIZE_4)
  {
    tile_a[ty][tx] = (row < M) ? a[row * K + i + tx] : 0;
    tile_b[ty][tx] = (col < N) ? b[col * K + i + ty] : 0;
    __syncthreads();
    ans += taxtb_4;
    __syncthreads();
  }

  tile_a[ty][tx] = (row < M && i + tx < K) ? a[row * K + i + tx] : 0;
  tile_b[ty][tx] = (col < N && i + ty < K) ? b[col * K + i + ty] : 0;
  __syncthreads();
  ans += taxtb_4;
  __syncthreads();

  ans *= alpha;
  if (row < M && col < N)
  {
    if (beta != 0)
      c[row * N + col] = c[row * N + col] * beta + ans;
    else
      c[row * N + col] = ans;
  }
}

template <typename Dtype>
__global__ void kernel_gemm_tt(const int M, const int N, const int K,
  const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
  Dtype *c) {
  __shared__ Dtype tile_a[BLOCK_SIZE_4][BLOCK_SIZE_4];
  __shared__ Dtype tile_b[BLOCK_SIZE_4][BLOCK_SIZE_4];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE_4 + ty;
  int col = blockIdx.x * BLOCK_SIZE_4 + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - BLOCK_SIZE_4; i += BLOCK_SIZE_4)
  {
    tile_a[ty][tx] = (row < M) ? a[(i + tx) * M + row] : 0;
    tile_b[ty][tx] = (col < N) ? b[col * K + i + ty] : 0; //
    __syncthreads(); 
    ans += taxtb_4;
    __syncthreads();
  }

  tile_a[ty][tx] = (row < M && i + tx < K) ? a[(i + tx) * M + row] : 0;
  tile_b[ty][tx] = (col < N && i + ty < K) ? b[col * K + i + ty] : 0; //
  __syncthreads();
  ans += taxtb_4;
  __syncthreads();

  ans *= alpha;
  if (row < M && col < N)
  {
    if (beta != 0)
      c[row * N + col] = c[row * N + col] * beta + ans;
    else
      c[row * N + col] = ans;
  }
}




// gemm kernel
// 4: 1.3
// 7: 2.7
// 8: 4
// 14: 4.15
// 16: 5.08, pragma unroll(4): 5.18, manually unroll: 5.25
// 18: 4.5
// 20: 4.5
// 32: 4.85
// cublas: 10.5
// gemm interface
void caffe_gpu_gemm(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  int grid_rows = (M + BLOCK_SIZE_4 - 1) / BLOCK_SIZE_4;
  int grid_cols = (N + BLOCK_SIZE_4 - 1) / BLOCK_SIZE_4;
  dim3 gridSize(grid_cols, grid_rows);
  dim3 blockSize(BLOCK_SIZE_4, BLOCK_SIZE_4);
  if (TransA == CblasNoTrans && TransB == CblasNoTrans)
    kernel_gemm_nn<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);
  else if (TransA != CblasNoTrans && TransB == CblasNoTrans)
    kernel_gemm_tn<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);
  else if (TransA == CblasNoTrans && TransB != CblasNoTrans)
    kernel_gemm_nt<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);
  else
    kernel_gemm_tt<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);

  // hipError_t er1 = hipPeekAtLastError();
  // CUDA_CHECK(er1);
}

}
#endif