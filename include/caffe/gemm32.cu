#include "hip/hip_runtime.h"
#ifndef _GEMM32_CU_
#define _GEMM32_CU_

namespace gemm32 {

#define BLOCK_SIZE_32 32
#define taxtb_32 (tile_a[ty][0]*tile_b[0][tx]+tile_a[ty][1]*tile_b[1][tx]+tile_a[ty][2]*tile_b[2][tx]+tile_a[ty][3]*tile_b[3][tx]+ \
  tile_a[ty][4]*tile_b[4][tx]+tile_a[ty][5]*tile_b[5][tx]+tile_a[ty][6]*tile_b[6][tx]+tile_a[ty][7]*tile_b[7][tx]+ \
  tile_a[ty][8]*tile_b[8][tx]+tile_a[ty][9]*tile_b[9][tx]+tile_a[ty][10]*tile_b[10][tx]+tile_a[ty][11]*tile_b[11][tx]+ \
  tile_a[ty][12]*tile_b[12][tx]+tile_a[ty][13]*tile_b[13][tx]+tile_a[ty][14]*tile_b[14][tx]+tile_a[ty][15]*tile_b[15][tx]+ \
  tile_a[ty][16]*tile_b[16][tx]+tile_a[ty][17]*tile_b[17][tx]+tile_a[ty][18]*tile_b[18][tx]+tile_a[ty][19]*tile_b[19][tx]+ \
  tile_a[ty][20]*tile_b[20][tx]+tile_a[ty][21]*tile_b[21][tx]+tile_a[ty][22]*tile_b[22][tx]+tile_a[ty][23]*tile_b[23][tx]+ \
  tile_a[ty][24]*tile_b[24][tx]+tile_a[ty][25]*tile_b[25][tx]+tile_a[ty][26]*tile_b[26][tx]+tile_a[ty][27]*tile_b[27][tx]+ \
  tile_a[ty][28]*tile_b[28][tx]+tile_a[ty][29]*tile_b[29][tx]+tile_a[ty][30]*tile_b[30][tx]+tile_a[ty][31]*tile_b[31][tx])


template <typename Dtype>
__global__ void kernel_gemm_nn(const int M, const int N, const int K,
    const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
    Dtype *c) {
  __shared__ Dtype tile_a[BLOCK_SIZE_32][BLOCK_SIZE_32];
  __shared__ Dtype tile_b[BLOCK_SIZE_32][BLOCK_SIZE_32];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE_32 + ty;
  int col = blockIdx.x * BLOCK_SIZE_32 + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - BLOCK_SIZE_32; i += BLOCK_SIZE_32)
  {
    tile_a[ty][tx] = (row < M) ? a[row * K + i + tx] : 0;
    tile_b[ty][tx] = (col < N) ? b[(i + ty) * N + col] : 0;
    __syncthreads();
    ans += taxtb_32;
    __syncthreads();
  }

  tile_a[ty][tx] = (row < M && i + tx < K) ? a[row * K + i + tx] : 0;
  tile_b[ty][tx] = (col < N && i + ty < K) ? b[(i + ty) * N + col] : 0;
  __syncthreads();
  ans += taxtb_32;
  __syncthreads();  

  ans *= alpha;
  if (row < M && col < N)
  {
    if (beta != 0)
      c[row * N + col] = c[row * N + col] * beta + ans;
    else
      c[row * N + col] = ans;
  }
}

template <typename Dtype>
__global__ void kernel_gemm_tn(const int M, const int N, const int K,
  const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
  Dtype *c) {
  __shared__ Dtype tile_a[BLOCK_SIZE_32][BLOCK_SIZE_32];
  __shared__ Dtype tile_b[BLOCK_SIZE_32][BLOCK_SIZE_32];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE_32 + ty;
  int col = blockIdx.x * BLOCK_SIZE_32 + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - BLOCK_SIZE_32; i += BLOCK_SIZE_32)
  {
    tile_a[ty][tx] = (row < M) ? a[(i + tx) * M + row] : 0;
    tile_b[ty][tx] = (col < N) ? b[(i + ty) * N + col] : 0;
    __syncthreads();
    ans += taxtb_32;
    __syncthreads();
  }

  tile_a[ty][tx] = (row < M && i + tx < K) ? a[(i + tx) * M + row] : 0;
  tile_b[ty][tx] = (col < N && i + ty < K) ? b[(i + ty) * N + col] : 0;
  __syncthreads();
  ans += taxtb_32;
  __syncthreads();

  ans *= alpha;
  if (row < M && col < N)
  {
    if (beta != 0)
      c[row * N + col] = c[row * N + col] * beta + ans;
    else
      c[row * N + col] = ans;
  }
}

template <typename Dtype>
__global__ void kernel_gemm_nt(const int M, const int N, const int K,
  const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
  Dtype *c) {
  __shared__ Dtype tile_a[BLOCK_SIZE_32][BLOCK_SIZE_32];
  __shared__ Dtype tile_b[BLOCK_SIZE_32][BLOCK_SIZE_32];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE_32 + ty;
  int col = blockIdx.x * BLOCK_SIZE_32 + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - BLOCK_SIZE_32; i += BLOCK_SIZE_32)
  {
    tile_a[ty][tx] = (row < M) ? a[row * K + i + tx] : 0;
    tile_b[ty][tx] = (col < N) ? b[col * K + i + ty] : 0;
    __syncthreads();
    ans += taxtb_32;
    __syncthreads();
  }

  tile_a[ty][tx] = (row < M && i + tx < K) ? a[row * K + i + tx] : 0;
  tile_b[ty][tx] = (col < N && i + ty < K) ? b[col * K + i + ty] : 0;
  __syncthreads();
  ans += taxtb_32;
  __syncthreads();

  ans *= alpha;
  if (row < M && col < N)
  {
    if (beta != 0)
      c[row * N + col] = c[row * N + col] * beta + ans;
    else
      c[row * N + col] = ans;
  }
}

template <typename Dtype>
__global__ void kernel_gemm_tt(const int M, const int N, const int K,
  const Dtype alpha, const Dtype *a, const Dtype *b, const Dtype beta,
  Dtype *c) {
  __shared__ Dtype tile_a[BLOCK_SIZE_32][BLOCK_SIZE_32];
  __shared__ Dtype tile_b[BLOCK_SIZE_32][BLOCK_SIZE_32];

  int ty = threadIdx.y;
  int tx = threadIdx.x;
  int row = blockIdx.y * BLOCK_SIZE_32 + ty;
  int col = blockIdx.x * BLOCK_SIZE_32 + tx;
  Dtype ans = 0;

  int i;
  for (i = 0; i < K - BLOCK_SIZE_32; i += BLOCK_SIZE_32)
  {
    tile_a[ty][tx] = (row < M) ? a[(i + tx) * M + row] : 0;
    tile_b[ty][tx] = (col < N) ? b[col * K + i + ty] : 0; //
    __syncthreads(); 
    ans += taxtb_32;
    __syncthreads();
  }

  tile_a[ty][tx] = (row < M && i + tx < K) ? a[(i + tx) * M + row] : 0;
  tile_b[ty][tx] = (col < N && i + ty < K) ? b[col * K + i + ty] : 0; //
  __syncthreads();
  ans += taxtb_32;
  __syncthreads();

  ans *= alpha;
  if (row < M && col < N)
  {
    if (beta != 0)
      c[row * N + col] = c[row * N + col] * beta + ans;
    else
      c[row * N + col] = ans;
  }
}




// gemm kernel
// 4: 1.3
// 7: 2.7
// 8: 4
// 14: 4.15
// 16: 5.08, pragma unroll(4): 5.18, manually unroll: 5.25
// 18: 4.5
// 20: 4.5
// 32: 4.85
// cublas: 10.5
// gemm interface
void caffe_gpu_gemm(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  int grid_rows = (M + BLOCK_SIZE_32 - 1) / BLOCK_SIZE_32;
  int grid_cols = (N + BLOCK_SIZE_32 - 1) / BLOCK_SIZE_32;
  dim3 gridSize(grid_cols, grid_rows);
  dim3 blockSize(BLOCK_SIZE_32, BLOCK_SIZE_32);
  if (TransA == CblasNoTrans && TransB == CblasNoTrans)
    kernel_gemm_nn<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);
  else if (TransA != CblasNoTrans && TransB == CblasNoTrans)
    kernel_gemm_tn<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);
  else if (TransA == CblasNoTrans && TransB != CblasNoTrans)
    kernel_gemm_nt<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);
  else
    kernel_gemm_tt<float><<<gridSize, blockSize>>>(M, N, K, alpha, A, B, beta, C);

  // hipError_t er1 = hipPeekAtLastError();
  // CUDA_CHECK(er1);
}

}
#endif